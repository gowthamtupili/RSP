#include "hip/hip_runtime.h"
#include "ego_estimation.cuh"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <numeric>
#include <cmath>

namespace EgoMotion {

    __global__ void extract_valid_relative_speeds(
        const TargetProcessing::CUDATarget* targets,
        double* validSpeeds,
        int* validCount,
        int numTargets
    ) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= numTargets) return;

        double speed = targets[idx].relativeSpeed;
        if (fabs(speed) > 0.1) {
            int insertIdx = atomicAdd(validCount, 1);
            validSpeeds[insertIdx] = speed;
        }
    }

    double estimate_ego_motion_gpu(const std::vector<TargetProcessing::CUDATarget>& h_targets) {
        if (h_targets.empty()) return 0.0;

        int numTargets = h_targets.size();
        TargetProcessing::CUDATarget* d_targets;
        double* d_validSpeeds;
        int* d_validCount;
        int maxValid = numTargets;

        hipMalloc(&d_targets, sizeof(TargetProcessing::CUDATarget) * numTargets);
        hipMemcpy(d_targets, h_targets.data(), sizeof(TargetProcessing::CUDATarget) * numTargets, hipMemcpyHostToDevice);
        hipMalloc(&d_validSpeeds, sizeof(double) * maxValid);
        hipMalloc(&d_validCount, sizeof(int));
        hipMemset(d_validCount, 0, sizeof(int));

        int threadsPerBlock = 256;
        int blocks = (numTargets + threadsPerBlock - 1) / threadsPerBlock;
        extract_valid_relative_speeds<<<blocks, threadsPerBlock>>>(
            d_targets, d_validSpeeds, d_validCount, numTargets);
        hipDeviceSynchronize();

        // Copy back valid count
        int h_validCount = 0;
        hipMemcpy(&h_validCount, d_validCount, sizeof(int), hipMemcpyDeviceToHost);
        if (h_validCount == 0) {
            hipFree(d_targets); hipFree(d_validSpeeds); hipFree(d_validCount);
            return 0.0;
        }

        std::vector<double> h_validSpeeds(h_validCount);
        hipMemcpy(h_validSpeeds.data(), d_validSpeeds, sizeof(double) * h_validCount, hipMemcpyDeviceToHost);

        double sum = std::accumulate(h_validSpeeds.begin(), h_validSpeeds.end(), 0.0);
        double avg = sum / h_validCount;

        hipFree(d_targets); hipFree(d_validSpeeds); hipFree(d_validCount);
        return avg;
    }

}
